#include <hip/hip_runtime.h>
#include "kernels.h"

float* sigmoidDispatcher(float* A_d, float* A_h, int N) {
	float* res_d = nullptr;

	hipMalloc(&A_d, sizeof(float) * N);
	hipMalloc(&res_d, sizeof(float) * N);

	hipMemcpy(A_d, A_h, sizeof(float) * N, hipMemcpyHostToDevice);

	dim3 blockDim(N);
	dim3 gridDim(1);

	sigmoidKernel<<<gridDim, blockDim>>>(A_d, res_d, N);	
	hipDeviceSynchronize();

	hipMemcpy(res_d, A_d, sizeof(float) * N, hipMemcpyDeviceToHost);
	hipFree(A_d);

	return res_h;
}
