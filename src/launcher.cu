#include <iostream>
#include "kernels.h"
#include <hip/hip_runtime.h>

// Pybind11 imports
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

namespace py = pybind11;

std::string LaunchSigmoid(py::array_t<float> A, int N) {
	py::buffer_info A_buff = A.request();
	
	if (A_buff.ndim > 1) {
		return "ndims cant be greater than 1. please ensure only vectors are passed";
	}

	if (A_buff.size > 256) {
		return "nope cant go beyond 256";
	}

	float* A_d = nullptr;
	float* A_h = static_cast<float*> (A_buff.ptr);

	hipMalloc(&A_d, sizeof(float) * A_buff.size);
	hipMemcpy(A_d, A_h, sizeof(float) * A_buff.size, hipMemcpyHostToDevice);

	dim3 blockDim(N);
	dim3 gridDim(1);

	sigmoidKernel<<<gridDim, blockDim>>>(A_d, N);	
	hipDeviceSynchronize();

	hipMemcpy(A_h, A_d, sizeof(float) * N, hipMemcpyDeviceToHost);
	hipFree(A_d);

	return "Success";
}

void LaunchCopyData(py::array_t<float> A, int N) {
}

PYBIND11_MODULE(cuPatcher, m) {
	m.doc() = "Basic CUDA dispatcher written by Rishik00";

	m.def("launch_sigmoid", &LaunchSigmoid, "A kernel that can do sigmoid using cuda");
	m.def("launch_copy", &LaunchCopyData, "A kernel that copies data");
}
