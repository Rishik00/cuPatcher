#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernels.h"

__global__ void dataCopy(float* A, float* B, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		B[idx] = A[idx];
	}
}

__global__ void sigmoidKernel(float* A, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		A[idx] = 1 / (1 + expf(A[idx]);
	}
}

__global__ void matMul (float* A, float* B, float* C, int N, int M) {

}

__global__ void RectMul (float* A, float* B, float* C, int N, int M, int K) {

}
