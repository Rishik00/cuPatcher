#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void dataCopy(float* A, float* B, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		B[idx] = A[idx];
	}
}

__global__ void sigmoidKernel(float* A, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N) {
		A[idx] = 1 / (1 + expf(A[idx]);
	}
}


__global__ void sumKernel(float* A, float* sum, int N) {
	

}



